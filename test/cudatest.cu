#include "hip/hip_runtime.h"
#include "cudatest.h"

#include <iostream>
#include <algorithm>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include "util.h"

inline void gpuAssert(hipError_t code, const char * file, int line, bool Abort=true) {
	if(code != hipSuccess) {
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),file,line);
		if (Abort) exit(code);
	}
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

__global__ void cuda_findOriginDistance(const unsigned int numElements, const float* pointList, float* centroid) {
	const int i = blockIdx.x*blockDim.x + threadIdx.x;

	__shared__ float sum[3];
	if(threadIdx.x == 0) {
		sum[0] = 0;
		sum[1] = 0;
		sum[2] = 0;
	}

	__syncthreads();
	if(i < numElements) {
		int sumIdx = i % 3;
		atomicAdd(&sum[sumIdx], pointList[i]);
	}
	__syncthreads();
	if(threadIdx.x == 0) {
		atomicAdd(&centroid[0], sum[0]);
		atomicAdd(&centroid[1], sum[1]);
		atomicAdd(&centroid[2], sum[2]);
	}

	__syncthreads();
	if(blockIdx.x == 0 && threadIdx.x == 0) {
		centroid[0] = centroid[0] / numElements;
		centroid[1] = centroid[1] / numElements;
		centroid[2] = centroid[2] / numElements;
	}
}

__global__ void cuda_translate(const unsigned int numElements, float* pointList, float* vec) {
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	int vecIdx = i % 3;

	pointList[i] -= vec[vecIdx];
}

__global__ void cuda_transpose(const unsigned int numElements, float* pointList, float* pointListTransposed) {
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
}

int cudaTest() {
	const unsigned int numElements1 = 8;
	const unsigned int numElements2 = 8;
	const unsigned int maxNumElements = std::max(numElements1, numElements2);

	const unsigned int numDimensions = 3;
	float* pointList1 = new float[numElements1*numDimensions];
	float* pointList2 = new float[numElements2*numDimensions];
// 2x1x1 cube
	pointList1[0 + 0] = 0; pointList1[0 + 1] = 0; pointList1[0 + 2] = 0;
	pointList1[3 + 0] = 2; pointList1[3 + 1] = 0; pointList1[3 + 2] = 0;
	pointList1[6 + 0] = 2; pointList1[6 + 1] = 1; pointList1[6 + 2] = 0;
	pointList1[9 + 0] = 0; pointList1[9 + 1] = 1; pointList1[9 + 2] = 0;
	pointList1[12 + 0] = 0; pointList1[12 + 1] = 0; pointList1[12 + 2] = 1;
	pointList1[15 + 0] = 2; pointList1[15 + 1] = 0; pointList1[15 + 2] = 1;
	pointList1[18 + 0] = 2; pointList1[18 + 1] = 1; pointList1[18 + 2] = 1;
	pointList1[21 + 0] = 0; pointList1[21 + 1] = 1; pointList1[21 + 2] = 1;

	// 1x1x2 cube
	pointList2[0 + 0] = 0; pointList2[0 + 1] = 0; pointList2[0 + 2] = 0;
	pointList2[3 + 0] = 0; pointList2[3 + 1] = 0; pointList2[3 + 2] = 2;
	pointList2[6 + 0] = 0; pointList2[6 + 1] = 1; pointList2[6 + 2] = 2;
	pointList2[9 + 0] = 0; pointList2[9 + 1] = 1; pointList2[9 + 2] = 0;
	pointList2[12 + 0] = -1; pointList2[12 + 1] = 0; pointList2[12 + 2] = 0;
	pointList2[15 + 0] = -1; pointList2[15 + 1] = 0; pointList2[15 + 2] = 2;
	pointList2[18 + 0] = -1; pointList2[18 + 1] = 1; pointList2[18 + 2] = 2;
	pointList2[21 + 0] = -1; pointList2[21 + 1] = 1; pointList2[21 + 2] = 0;

	hipSetDevice(0);

	size_t bytes1 = numElements1*sizeof(float);
	size_t bytes2 = numElements2*sizeof(float);

	float* d_pointList1;
	float* d_pointList2;
	size_t bytesCentroid = numDimensions*sizeof(float*);
	float* d_centroid1;
	float* d_centroid2;
	float centroid1[3] = {0,0,0};
	float centroid2[3] = {0,0,0};
	gpuErrchk(hipMalloc(&d_pointList1, bytes1));
	gpuErrchk(hipMalloc(&d_pointList2, bytes2));

	gpuErrchk(hipMemcpy(d_pointList1, pointList1, bytes1, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_pointList2, pointList2, bytes2, hipMemcpyHostToDevice));

	gpuErrchk(hipMalloc(&d_centroid1, bytesCentroid));
	gpuErrchk(hipMalloc(&d_centroid2, bytesCentroid));

	gpuErrchk(hipMemset(d_centroid1, 0, bytesCentroid));
	gpuErrchk(hipMemset(d_centroid2, 0, bytesCentroid));

	int blockSize, gridSize;

	blockSize = 1024;
	gridSize = (int)ceil((float)maxNumElements/blockSize);

	cuda_findOriginDistance<<<gridSize, blockSize>>>(numElements1, d_pointList1, d_centroid1);
	cuda_findOriginDistance<<<gridSize, blockSize>>>(numElements2, d_pointList2, d_centroid2);

	hipMemcpy(centroid1, d_centroid1, bytesCentroid, hipMemcpyDeviceToHost);
	hipMemcpy(centroid2, d_centroid2, bytesCentroid, hipMemcpyDeviceToHost);

	std::cout << "CUDA:" << std::endl;
	std::cout << centroid1[0] << " " << centroid1[1] << " " << centroid1[2] << std::endl;
	std::cout << centroid2[0] << " " << centroid2[1] << " " << centroid2[2] << std::endl;

	cuda_translate<<<gridSize, blockSize>>>(numElements1, d_pointList1, d_centroid1);
	cuda_translate<<<gridSize, blockSize>>>(numElements2, d_pointList2, d_centroid1);

	hipMemcpy(pointList1, d_pointList1, bytesCentroid, hipMemcpyDeviceToHost);
	hipMemcpy(pointList2, d_pointList2, bytesCentroid, hipMemcpyDeviceToHost);

	std::cout << "CUDA (after translation):" << std::endl;
	std::cout << "First:" << std::endl;
	printMatrix(numElements1, numDimensions, pointList1);
	std::cout << "Second:" << std::endl;
	printMatrix(numElements2, numDimensions, pointList2);

	float* d_pointList1Transposed;
	gpuErrchk(hipMalloc(&d_pointList1Transposed, bytes1));
	gpuErrchk(hipMemset(d_pointList1Transposed, 0, bytes1));

	cuda_transpose<<<gridSize, blockSize>>>(numElements1, d_pointList1, d_pointList1Transposed);
	
	hipFree(d_pointList1Transposed);
	
	hipFree(d_centroid1);
	hipFree(d_centroid2);

	hipFree(d_pointList1);
	hipFree(d_pointList2);

	return 0;
}
