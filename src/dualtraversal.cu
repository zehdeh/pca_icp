#include "hip/hip_runtime.h"
#include "dualtraversal.h"
#include <stack>
#include <queue>

#include <nvbio/basic/priority_queue.h>
#include <nvbio/basic/vector_view.h>

#define INFTY 9999

struct CudaPoint : Point {
	__host__ __device__ CudaPoint(const Point& p) {
		x = p.x;
		y = p.y;
		z = p.z;
	}

	__host__ __device__ float operator[](const unsigned int index) const
	{
		return *((float*) this + index);
	}

	__host__ __device__ float & operator[](const unsigned int index)
	{
		return *((float*) this + index);
	}

	__host__ __device__ CudaPoint operator-(const Point& other) const
	{
		Point result;
		result.x = x - other.x;
		result.y = y - other.y;
		result.z = z - other.z;

		return result;
	}

	__host__ __device__ float length() const
	{
		return sqrtf(x * x + y * y + z * z);
	}
};

struct workItem {
	workItem(const unsigned int queryNodeIdx,
		const unsigned int nodeIdx) : queryNodeIdx(queryNodeIdx), nodeIdx(nodeIdx) {}

	unsigned int queryNodeIdx;
	unsigned int nodeIdx;
};

struct prioritizedWorkItem : workItem {
	prioritizedWorkItem(const unsigned int queryNodeIdx,
		const unsigned int nodeIdx, const float priority) : workItem(queryNodeIdx, nodeIdx), priority(priority) {}
	float priority;
};

bool operator<(const prioritizedWorkItem& lhs, const prioritizedWorkItem& rhs) {
	return lhs.priority < rhs.priority;
}

struct PriorityComparator {
	bool operator()(const prioritizedWorkItem& lhs, const prioritizedWorkItem& rhs) {
		return lhs.priority < rhs.priority;
	}
};

struct PointIndexComparator {
	char splitDim;
	const std::vector<Point> &points;
	const std::vector<unsigned int> &pointIndices;

	PointIndexComparator(const char splitDim, const std::vector<Point> &points, const std::vector<unsigned int> &pointIndices) :
		splitDim(splitDim), points(points), pointIndices(pointIndices) {
		}

	bool operator()(const unsigned int &i, const unsigned int &j)
	{
		return (((const CudaPoint)points[i])[splitDim] < ((const CudaPoint)points[j])[splitDim]);
	}
};

unsigned int makeKdLeafTree(const std::vector<Point>& points, std::vector<unsigned int> pointIndices, std::vector<KdNode2>& nodes,
	const char splitDim, const unsigned int currentNodeIdx, const int parentIdx) {
	KdNode2& currentNode = nodes[currentNodeIdx];
	currentNode.parentIdx = parentIdx;

	if(pointIndices.size() == 1) {
		currentNode.pointIdx = pointIndices[0];
		currentNode.isLeaf = true;
		for(unsigned int i = 0; i < 3; i++) {
			currentNode.boundaries[i].first = currentNode.boundaries[i].second = ((CudaPoint)points[currentNode.pointIdx])[i];
		}
	/*} else if(pointIndices.size() == 2) {
		currentNode.leftChild = (((CudaPoint)points[pointIndices[0]])[splitDim]<((CudaPoint)points[pointIndices[1]])[splitDim])?pointIndices[0]:pointIndices[1];
		currentNode.rightChild = (((CudaPoint)points[pointIndices[0]])[splitDim]>((CudaPoint)points[pointIndices[1]])[splitDim])?pointIndices[0]:pointIndices[1];
		
		for(unsigned int i = 0; i < 3; i++) {
			currentNode.boundaries[i].first = std::min(((CudaPoint)points[pointIndices[0]])[i],((CudaPoint)points[pointIndices[1]])[i]);
			currentNode.boundaries[i].second = std::max(((CudaPoint)points[pointIndices[0]])[i],((CudaPoint)points[pointIndices[1]])[i]);
		}

		currentNode.isLeaf = true;*/
	} else {
		sort(pointIndices.begin(), pointIndices.end(), PointIndexComparator(splitDim, points, pointIndices));

		unsigned int splitIndex = pointIndices.size() / 2;
		char newSplitDim = (splitDim + 1) % 3;
		std::vector<unsigned int> pointIndicesLeft(pointIndices.begin(), pointIndices.begin() + splitIndex);
		makeKdLeafTree(points, pointIndicesLeft, nodes, newSplitDim, currentNodeIdx + 1, currentNodeIdx);

		std::vector<unsigned int> pointIndicesRight(pointIndices.begin() + splitIndex, pointIndices.end());
		currentNode.rightChild = makeKdLeafTree(points, pointIndicesRight, nodes, newSplitDim, currentNodeIdx + (2*pointIndicesLeft.size()), currentNodeIdx);

		for(unsigned int i = 0; i < 3; i++) {
			currentNode.boundaries[i].first = std::min(nodes[currentNodeIdx + 1].boundaries[i].first,nodes[currentNode.rightChild].boundaries[i].first);
			currentNode.boundaries[i].second = std::max(nodes[currentNodeIdx + 1].boundaries[i].second,nodes[currentNode.rightChild].boundaries[i].second);
		}
	}

	return currentNodeIdx;
}

std::vector<KdNode2> makeKdLeafTree(const std::vector<Point>& points) {
	std::vector<unsigned int> pointIndices(points.size());
	std::vector<KdNode2> nodes(2*points.size()-1);
	for(unsigned int i = 0; i < points.size(); i++) {
		pointIndices[i] = i;
	}
	const unsigned int rootIdx = makeKdLeafTree(points, pointIndices, nodes, KdNode::X, 0, -1);
#ifdef VERBOSE
	nodes[rootIdx].print(nodes, points, 0, rootIdx);
#endif

	return nodes;
}

float maxDescendantDistance(const std::vector<KdNode2>& queryNodes, const unsigned int nodeIdx, const float* const distances) {
	const KdNode2& node = queryNodes[nodeIdx];
	if(node.isLeaf) {
		return distances[node.pointIdx];
	} else {
		return std::max(maxDescendantDistance(queryNodes, nodeIdx + 1, distances), maxDescendantDistance(queryNodes, node.rightChild, distances));
	}
}

float minNodeDistance(const KdNode2& query, const KdNode2& node) {
	float distance = 0;
	for(int i = 0; i < 3; i++) {
		float qMin = std::min(query.boundaries[i].first, query.boundaries[i].second);
		float qMax = std::max(query.boundaries[i].first, query.boundaries[i].second);
		float nMin = std::min(node.boundaries[i].first, node.boundaries[i].second);
		float nMax = std::max(node.boundaries[i].first, node.boundaries[i].second);
		if(qMax < nMin) {
			float d = nMin - qMax;
			distance += d*d;
		} else if(nMax < qMin){
			float d = qMin - nMax;
			distance += d*d;
		}
	}
	return sqrtf(distance);
}

void printChildren(const std::vector<KdNode2>& queryNodes, const unsigned int nodeIdx, const float* const distances) {
	const KdNode2& node = queryNodes[nodeIdx];
	if(node.isLeaf) {
		std::cout << node.pointIdx << " ";
	} else {
		printChildren(queryNodes, nodeIdx + 1, distances);
		printChildren(queryNodes, node.rightChild, distances);
	}
}

void dualBaseCase(const CudaPoint& query, const CudaPoint& point,
	const unsigned int currentQueryIdx, const unsigned int currentPointIdx,
	unsigned int* Nns, float* distances) {

	const float distance = (point - query).length();
	if(distance < distances[currentQueryIdx]) {
		Nns[currentQueryIdx] = currentPointIdx;
		distances[currentQueryIdx] = distance;
	}
}

void dualTreeStep(const std::vector<KdNode2>& nodes, const std::vector<KdNode2>& queryNodes,
	const std::vector<Point>& points, const std::vector<Point>& queries,
	const unsigned int currentNodeIdx, const unsigned int currentQueryNodeIdx,
	unsigned int* Nns, float* distances,
	std::stack< workItem >& stack) {
#ifdef VERBOSE
	std::cout << "q: " << currentQueryNodeIdx << " n: " << currentNodeIdx << std::endl;
#endif

	const KdNode2& currentQueryNode = queryNodes[currentQueryNodeIdx];
	const KdNode2& currentNode = nodes[currentNodeIdx];
	
	const float nodeDistance = minNodeDistance(currentQueryNode, currentNode);
	const float maxDescendant = maxDescendantDistance(queryNodes, currentQueryNodeIdx, distances);
	if(nodeDistance > maxDescendant) {
#ifdef VERBOSE
		std::cout << "Pruning query node " << currentQueryNodeIdx << (currentQueryNode.isLeaf?"(leaf)":"") 
			<< " with node " << currentNodeIdx << (currentNode.isLeaf?"(leaf)":"") << " (" << nodeDistance << " > " << maxDescendant << ")" << std::endl;
		std::cout << "Skipping comparison of queries " << std::endl;
		printChildren(queryNodes, currentQueryNodeIdx, distances);
		std::cout << std::endl << " with points " << std::endl;
		printChildren(nodes, currentNodeIdx, distances);
		std::cout << std::endl;
#endif
		return;
	}

	if(currentQueryNode.isLeaf && currentNode.isLeaf) {
#ifdef VERBOSE
		std::cout << "Comparing " << currentQueryNode.pointIdx << " with " << currentNode.pointIdx << std::endl;
#endif
		dualBaseCase(queries[currentQueryNode.pointIdx], points[currentNode.pointIdx], 
			currentQueryNode.pointIdx, currentNode.pointIdx,
			Nns, distances);
	} else if(currentQueryNode.isLeaf && !currentNode.isLeaf) {
#ifdef VERBOSE
		std::cout << "pushing (" << currentQueryNodeIdx << "," << (currentNodeIdx + 1) << ")" << std::endl;
		std::cout << "pushing (" << currentQueryNodeIdx << "," << currentNode.rightChild << ")" << std::endl;
#endif
		stack.push(workItem(currentQueryNodeIdx,currentNodeIdx + 1));
		stack.push(workItem(currentQueryNodeIdx,currentNode.rightChild));
	} else if(!currentQueryNode.isLeaf && currentNode.isLeaf) {
#ifdef VERBOSE
		std::cout << "pushing (" << (currentQueryNodeIdx + 1) << "," << currentNodeIdx << ")" << std::endl;
		std::cout << "pushing (" << currentQueryNode.rightChild << "," << currentNodeIdx << ")" << std::endl;
#endif
		stack.push(workItem(currentQueryNodeIdx + 1,currentNodeIdx));
		stack.push(workItem(currentQueryNode.rightChild,currentNodeIdx));
	} else {
#ifdef VERBOSE
		std::cout << "pushing (" << (currentQueryNodeIdx + 1) << "," << (currentNodeIdx + 1) << ")" << std::endl;
		std::cout << "pushing (" << currentQueryNode.rightChild << "," << currentNode.rightChild << ")" << std::endl;
		std::cout << "pushing (" << (currentQueryNodeIdx + 1) << "," << currentNode.rightChild << ")" << std::endl;
		std::cout << "pushing (" << currentQueryNode.rightChild << "," << currentNodeIdx + 1 << ")" << std::endl;
#endif
		stack.push(workItem(currentQueryNodeIdx + 1,currentNodeIdx + 1));
		stack.push(workItem(currentQueryNodeIdx + 1,currentNode.rightChild));
		stack.push(workItem(currentQueryNode.rightChild,currentNodeIdx + 1));
		stack.push(workItem(currentQueryNode.rightChild,currentNode.rightChild));
	}
}

void cpu_findNnDual(const std::vector<KdNode2>& nodes, const std::vector<KdNode2>& queryNodes,
		const std::vector<Point>& points, const std::vector<Point>& queries,
		std::vector<int>& results) {
	
	unsigned int Nns[queries.size()];
	float distances[queries.size()];
	std::stack< workItem > stack;
	memset(Nns, 0, sizeof(Nns));
	for(unsigned int i = 0; i < queries.size(); i++) {
		distances[i] = INFTY;
	}

	stack.push(workItem(0, 0));
	while(!stack.empty()) {
		workItem work = stack.top();
		stack.pop();
		dualTreeStep(nodes, queryNodes, points, queries, work.nodeIdx, work.queryNodeIdx, Nns, distances, stack);
	}
	results.insert(results.begin(), &Nns[0], &Nns[sizeof(Nns) / sizeof(unsigned int)]);
}

float score(const std::vector<KdNode2>& queryNodes,
	const unsigned int currentQueryNodeIdx, const KdNode2& currentNode,
	const float* distances) {

	const KdNode2& currentQueryNode = queryNodes[currentQueryNodeIdx];

	const float nodeDistance = minNodeDistance(currentQueryNode, currentNode);
	const float maxDescendant = maxDescendantDistance(queryNodes, currentQueryNodeIdx, distances);
	if(nodeDistance < maxDescendant) {
		return maxDescendant - nodeDistance;
	}

	return INFTY;
}

void dualTreeStepPrioritized(const std::vector<KdNode2>& nodes, const std::vector<KdNode2>& queryNodes,
	const std::vector<Point>& points, const std::vector<Point>& queries,
	const unsigned int currentNodeIdx, const unsigned int currentQueryNodeIdx,
	unsigned int* Nns, float* distances,
	std::priority_queue<prioritizedWorkItem> stack) {

	const KdNode2& currentQueryNode = queryNodes[currentQueryNodeIdx];
	const KdNode2& currentNode = nodes[currentNodeIdx];
	
	float s = 0;
	if(currentQueryNode.isLeaf && currentNode.isLeaf) {
		dualBaseCase(queries[currentQueryNode.pointIdx], points[currentNode.pointIdx], 
			currentQueryNode.pointIdx, currentNode.pointIdx,
			Nns, distances);
	} else if(currentQueryNode.isLeaf && !currentNode.isLeaf) {
		s = score(queryNodes, currentQueryNodeIdx, nodes[currentNodeIdx + 1], distances);
		if(s < INFTY) {
			stack.push(prioritizedWorkItem(currentQueryNodeIdx,currentNodeIdx + 1,1/s));
		}
		s = score(queryNodes, currentQueryNodeIdx, nodes[currentNode.rightChild], distances);
		if(s < INFTY) {
			stack.push(prioritizedWorkItem(currentQueryNodeIdx,currentNode.rightChild,1/s));
		}
	} else if(!currentQueryNode.isLeaf && currentNode.isLeaf) {
		s = score(queryNodes, currentQueryNodeIdx + 1, currentNode, distances);
		if(s < INFTY) {
			stack.push(prioritizedWorkItem(currentQueryNodeIdx + 1,currentNodeIdx, 1/s));
		}
		s = score(queryNodes, currentQueryNode.rightChild, currentNode, distances);
		if(s < INFTY) {
			stack.push(prioritizedWorkItem(currentQueryNode.rightChild,currentNodeIdx, 1/s));
		}
	} else {
		s = score(queryNodes, currentQueryNodeIdx + 1, nodes[currentNodeIdx + 1], distances);
		if(s < INFTY) {
			stack.push(prioritizedWorkItem(currentQueryNodeIdx + 1,currentNodeIdx + 1,1/s));
		}
		s = score(queryNodes, currentQueryNodeIdx + 1, nodes[currentNode.rightChild], distances);
		if(s < INFTY) {
			stack.push(prioritizedWorkItem(currentQueryNodeIdx + 1,currentNode.rightChild,1/s));
		}
		s = score(queryNodes, currentQueryNode.rightChild, nodes[currentNodeIdx + 1], distances);
		if(s < INFTY) {
			stack.push(prioritizedWorkItem(currentQueryNode.rightChild,currentNodeIdx + 1,1/s));
		}
		s = score(queryNodes, currentQueryNode.rightChild, nodes[currentNode.rightChild], distances);
		if(s < INFTY) {
			stack.push(prioritizedWorkItem(currentQueryNode.rightChild,currentNode.rightChild,1/s));
		}
	}
}

void cpu_findNnDualPrioritized(const std::vector<KdNode2>& nodes, const std::vector<KdNode2>& queryNodes,
		const std::vector<Point>& points, const std::vector<Point>& queries,
		std::vector<int>& results) {
	
	nvbio::vector_view<prioritizedWorkItem*> priorityVector;
	nvbio::priority_queue<prioritizedWorkItem, nvbio::vector_view<prioritizedWorkItem*>, PriorityComparator > test(priorityVector);
	
	unsigned int Nns[queries.size()];
	float distances[queries.size()];
	std::priority_queue<prioritizedWorkItem> stack;
	memset(Nns, 0, sizeof(Nns));
	for(unsigned int i = 0; i < queries.size(); i++) {
		distances[i] = INFTY;
	}

	stack.push(prioritizedWorkItem(0, 0, 0));
	while(!stack.empty()) {
		prioritizedWorkItem work = stack.top();
		stack.pop();
		dualTreeStepPrioritized(nodes, queryNodes, points, queries, work.nodeIdx, work.queryNodeIdx, Nns, distances, stack);
	}
	results.insert(results.begin(), &Nns[0], &Nns[sizeof(Nns) / sizeof(unsigned int)]);
}

void cuda_findNnDual(const std::vector<KdNode2>& nodes, const std::vector<KdNode2>& queryNodes,
		const std::vector<Point>& points, const std::vector<Point>& queries,
		std::vector<int>& results) {
	
	Point* gPoints;
	KdNode2* gNodes;

	Point* gQueries;
	KdNode2* gQueryNodes;

	int* gNns;


	hipMalloc(&gPoints, sizeof(Point) * points.size());
	hipMalloc(&gNodes, sizeof(KdNode2) * nodes.size());
	hipMalloc(&gQueries, sizeof(Point) * queries.size());
	hipMalloc(&gQueryNodes, sizeof(KdNode2) * queryNodes.size());
	hipMalloc(&gNns, queries.size() * sizeof(int));

	hipMemcpy(gPoints, &(points[0]), sizeof(Point) * points.size(), hipMemcpyHostToDevice);
	hipMemcpy(gNodes, &(nodes[0]), sizeof(KdNode2) * nodes.size(), hipMemcpyHostToDevice);
	hipMemcpy(gQueries, &(queries[0]), sizeof(KdNode2) * queries.size(), hipMemcpyHostToDevice);
	hipMemcpy(gQueryNodes, &(queryNodes[0]), sizeof(KdNode2) * queryNodes.size(), hipMemcpyHostToDevice);

	hipFree(gPoints);
	hipFree(gNodes);
	hipFree(gQueries);
	hipFree(gNns);
}
