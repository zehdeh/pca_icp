#include "hip/hip_runtime.h"
#include "cuda_covariance.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include "cuda_util.h"

__global__ void kernel_findOriginDistance(const unsigned int numElements, const unsigned int numDimensions, const float* const pointList, float* const centroid);
__global__ void kernel_translate(const unsigned int numElements, const unsigned int numDimensions, float* pointList, float* vec);
__global__ void kernel_transpose(const unsigned int numElements, const unsigned int numDimensions, const float* const pointList, float* const pointListTransposed);
__global__ void kernel_findCovariance(const unsigned int numElements, const unsigned int numDimensions, const float* const pointList1, const float* const pointList2, float* const covariance);

//FIXME: Do not rely on static variables here
float* d_pointList1;
float* d_pointList2;

float** getDevicePointList1() {
	return &d_pointList1;
}
float** getDevicePointList2() {
	return &d_pointList2;
}

void cuda_initPointLists(const unsigned int numElements, const unsigned int numDimensions, const float* const pointList1, const float* const pointList2) {
	size_t bytes = numDimensions*numElements*sizeof(float);

	gpuErrchk(hipSetDevice(0));

	gpuErrchk(hipMalloc(&d_pointList1, bytes));
	gpuErrchk(hipMemcpy(d_pointList1, pointList1, bytes, hipMemcpyHostToDevice));

	gpuErrchk(hipMalloc(&d_pointList2, bytes));
	gpuErrchk(hipMemcpy(d_pointList2, pointList2, bytes, hipMemcpyHostToDevice));
}
void cuda_destroyPointList(float* d_pointList) {
	hipFree(d_pointList);
}

void cuda_downloadPointList(const unsigned numElements, const unsigned int numDimensions, float* const pointList, float* d_pointList) {
	size_t bytes = numDimensions*numElements*sizeof(float);

	gpuErrchk(hipMemcpy(pointList, d_pointList, bytes, hipMemcpyDeviceToHost));
}

void cuda_findOriginDistance(const unsigned int numElements, const unsigned int numDimensions, const float* const d_pointList, float* centroid) {
	size_t bytesCentroid = numDimensions*sizeof(float);
	float* d_centroid;

	gpuErrchk(hipMalloc(&d_centroid, bytesCentroid));
	gpuErrchk(hipMemset(d_centroid, 0, bytesCentroid));

	int blockSize, gridSize;
	blockSize = 1024;
	gridSize = (int)ceil((float)numDimensions*numElements/blockSize);

	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	kernel_findOriginDistance<<<gridSize, blockSize>>>(numElements, numDimensions, d_pointList, d_centroid);
	hipEventRecord(stop);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	std::cout << "Computing centroid took " << milliseconds << " ms" << std::endl;

	gpuErrchk(hipMemcpy(centroid, d_centroid, bytesCentroid, hipMemcpyDeviceToHost));
	gpuErrchk(hipFree(d_centroid));
}

void cuda_translate(const unsigned int numElements, const unsigned int numDimensions, float* const d_pointList, float* const centroid) {
	size_t bytesCentroid = numDimensions*sizeof(float);
	float* d_centroid;

	gpuErrchk(hipMalloc(&d_centroid, bytesCentroid));
	gpuErrchk(hipMemcpy(d_centroid, centroid, bytesCentroid, hipMemcpyHostToDevice));

	int blockSize, gridSize;
	blockSize = 1024;
	gridSize = (int)ceil((float)numDimensions*numElements/blockSize);

	kernel_translate<<<gridSize, blockSize>>>(numElements, numDimensions, d_pointList, d_centroid);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );

	//gpuErrchk(hipMemcpy(centroid, d_centroid, bytesCentroid, hipMemcpyDeviceToHost));
	gpuErrchk(hipFree(d_centroid));
}

void cuda_findCovariance(const unsigned int numElements, const unsigned int numDimensions, const float* const d_pointList1, const float* const d_pointList2, float* const covariance) {
	float* d_covariance;
	size_t bytesCovariance = sizeof(float)*numDimensions*numDimensions;

	gpuErrchk(hipMalloc(&d_covariance, bytesCovariance));
	gpuErrchk(hipMemset(d_covariance, 0, bytesCovariance));

	int blockSize, gridSize;
	blockSize = 1024;
	gridSize = (int)ceil((float)numDimensions*numDimensions*numElements/blockSize);
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	kernel_findCovariance<<<gridSize, blockSize>>>(numElements, numDimensions, d_pointList1, d_pointList2, d_covariance);
	hipEventRecord(stop);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	std::cout << "Computing covariance took " << milliseconds << " ms" << std::endl;

	gpuErrchk(hipMemcpy(covariance, d_covariance, bytesCovariance, hipMemcpyDeviceToHost));
	gpuErrchk(hipFree(d_covariance));
}

__global__ void kernel_findOriginDistance(const unsigned int numElements, const unsigned int numDimensions, const float* const pointList, float* const centroid) {
	const int i = blockIdx.x*blockDim.x + threadIdx.x;

/*
	if(i < numElements * numDimensions) {
		int sumIdx = i % 3;
		atomicAdd(&centroid[sumIdx], pointList[i]);
	}
	*/

	__shared__ float sum[3];
	if(threadIdx.x == 0) {
		sum[0] = 0;
		sum[1] = 0;
		sum[2] = 0;
	}

	__syncthreads();
	if(i < numElements * numDimensions) {
		int sumIdx = i % 3;
		atomicAdd(&sum[sumIdx], pointList[i] / numElements);
	}
	__syncthreads();
	if(threadIdx.x == 0) {
		atomicAdd(&centroid[0], sum[0]);
		atomicAdd(&centroid[1], sum[1]);
		atomicAdd(&centroid[2], sum[2]);
	}
}

__global__ void kernel_translate(const unsigned int numElements, const unsigned int numDimensions, float* pointList, float* vec) {
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i < numDimensions*numElements) {
		int vecIdx = i % numDimensions;

		pointList[i] -= vec[vecIdx];
	}
}

//FIXME: Not used anymore!
__global__ void kernel_transpose(const unsigned int numElements, const unsigned int numDimensions, const float* const pointList, float* const pointListTransposed) {
	const unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i < numDimensions*numElements) {
		const unsigned int j = i / numElements;
		const unsigned int k = i % numElements;

		pointListTransposed[i] = pointList[k*3 + j];
	}
}

__global__ void kernel_findCovariance(const unsigned int numElements, const unsigned int numDimensions, const float* const pointList1, const float* const pointList2, float* const covariance) {
	const unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

	if(i < numDimensions*numDimensions*numElements) {

		const unsigned int cov_j = i % numElements;
		const unsigned int cov_i = i / numElements;
		const unsigned int x = cov_i % numDimensions;
		const unsigned int y = cov_i / numDimensions;

		__syncthreads();

		const float elem = pointList1[y+cov_j*numDimensions]*pointList2[x + cov_j*numDimensions];
		atomicAdd(&covariance[y*numDimensions + x], elem);
		
		__syncthreads();
	}
}
